#include "hip/hip_runtime.h"
#include "model.h"
#include "params.h"
#include "reader.h"
#include <cmath>
#include <thread>

#include "shortcuts.h"


/**
 * Declare model static variables
 */
Model::Layer* Model::layers;
int Model::current_layer_count;


/**
 * Initialize the general model
 */
void Model::init () {
    layers = new Layer [LAYERS];
    current_layer_count = 0;
}


/**
 * Add a new layer to the model
 * Takes number of neurons in this layers along with their weights and biases
 */
void Model::add_layer (int neuron_count, double* weights, double* biases) {
    Layer& c_layer = layers [current_layer_count];
    c_layer.neuron_count = neuron_count;
    c_layer.input_count = current_layer_count > 0 ? layers [current_layer_count - 1].neuron_count : INPUT;
    current_layer_count ++;
    
    int b_weights = sizeof (double) * c_layer.input_count * c_layer.neuron_count;
    int b_biases = sizeof (double) * c_layer.neuron_count;
    hipMalloc (&(c_layer.weights), b_weights);
    hipMalloc (&(c_layer.biases), b_biases);
    hipMemcpy (c_layer.weights, weights, b_weights, hipMemcpyHostToDevice);
    hipMemcpy (c_layer.biases, biases, b_biases, hipMemcpyHostToDevice);
}


/**
 * Free memory taken by the general model
 */
void Model::free () {
    for (int i = 0; i < LAYERS; i ++) {
        hipFree (layers [i].weights);
        hipFree (layers [i].biases);
    }
    delete[] layers;
}


/** */
__global__ void reset_layer_gpu (int n, double* layer) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        data [id] = 0;
    }
}


/** */
__global__ void process_gpu (
    int input_count, double* inputs,
    int neuron_count, double* weights, double* biases,
    double* outputs
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        outputs [id] = (data [id] > 0);
    }
}


/**
 * Layer processing
 * Processes the given layer by taking an input to it
*/
void Model::process (int layer) {
    // Locate current layer, its inputs and outputs
    Layer& c_layer = layers [layer];
    double* input = data [layer];
    double* c_data = data [layer + 1];
    
    // For each neurone in this layer from each batch
    for (int u = 0; u < batch_size; u ++) {
        for (int i = 0; i < c_layer.neuron_count; i ++) {
            double& c_out = c_data [u * c_layer.neuron_count + i];

            // Compute the output for current neurone
            c_out = 0;
            for (int j = 0; j < c_layer.input_count; j ++) {
                c_out += input [u * c_layer.input_count + j] * c_layer.weights [i * c_layer.input_count + j];
            }
            c_out += c_layer.biases [i];
        }
    }
}


/** */
__global__ void relu_gpu (int n, double* data) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        data [id] *= (data [id] > 0);
    }
}


/**
 * Activate the given layer using ReLU
 */
void Model::relu (int layer) {
    double* c_data = data [layer + 1];
    int total = layers [layer].neuron_count * batch_size;
    for (int i = 0; i < total; i ++) {
        c_data [i] *= (c_data [i] > 0);
    }
}


/** */
__global__ softmax_gpu () {
    //
}


/**
 * Activate the last layer using softmax
 */
void Model::softmax () {
    // Locate the last layer and its outputs
    int categories = layers [LAYERS - 1].neuron_count;
    double* c_data = data [LAYERS];

    // For each output in last layer from each batch
    for (int u = 0; u < batch_size; u ++) {
        double exp_sum = 0;

        // Calculate exp sum
        for (int i = 0; i < categories; i ++) {
            exp_sum += exp (c_data [u * categories + i]);
        }

        // Calculate output
        double max = 0;
        for (int i = 0; i < categories; i ++) {
            c_data [i] = exp (c_data [u * categories + i]) / exp_sum;
            if (c_data [i] > max) {
                max = c_data [i];
                outputs [u] = i;
            }
        }
    }
}


/**
 * Model constructor
 * Takes a batch size as a parameter
 */
Model::Model (int batch_size) {
    this -> batch_size = batch_size;
    this -> current_input = 0;
    this -> ready = 0;
    data = new double* [LAYERS + 1];
    hipMalloc (&(data [0]), sizeof (double) * INPUT * batch_size);
    input = new double [INPUT * batch_size];
    for (int i = 1; i <= LAYERS; i ++) {
        hipMalloc (&(data [i]), sizeof (double) * layers [i - 1].neuron_count * batch_size);
    }
    outputs = new int [batch_size];
    mappings = new int [batch_size];
}


/**
 * Return true if all inputs are covered, otherwise false
 */ 
bool Model::is_ready () {
    return current_input == batch_size;
}


/**
 * Read tensor into current input array
 * Takes tensor filename, a position for mapping, and a pointer to the number of free readers
 */
void Model::process_input (const std::string& filename, int pos, std::atomic_int* free_readers) {
    mappings [current_input] = pos;
    std::thread t (read_input, filename, input + current_input * INPUT, &ready, free_readers);
    t.detach ();
    (*free_readers) --;
    current_input ++;
}


/**
 * Forward pass
 * Takes an auxiliary array to store results in,
 * a queue of models to make itself available again,
 * and a sub-batch parameter (optional)
*/
void Model::forward_pass (char* aux, tq* models, int sub_batch) {
    // Save original batch size
    int original_batch = batch_size;
    if (sub_batch > 0 && sub_batch < batch_size) {
        batch_size = sub_batch;
    }

    // Wait for all inputs to be read
    while (ready != batch_size) {IDLE}

    // Copy inputs to device
    hipMemcpy (data [0], input, sizeof (double) * INPUT * batch_size, hipMemcpyHostToDevice);

    // Activate layer K-1, then process it to layer K
    for (int i = 0; i < LAYERS - 1; i ++) {
        process (i);
        relu (i);
    }

    // Process and activate the last layer and get outputs
    process (LAYERS - 1);
    softmax ();

    // Write results to the auxilary output array
    for (int i = 0; i < batch_size; i ++) {
        int res = outputs [i];
        aux [mappings [i]] = res % 2 ? char (97 + res / 2) : char (65 + res / 2);
    }

    // Restore original batch size, current input and ready count
    batch_size = original_batch;
    current_input = 0;
    ready = 0;

    // Make this model available again by adding it to the queue of available models
    models -> push (this);
}


/**
 * Model destructor
 * Frees all model outputs and data
 */
Model::~Model () {
    delete[] input;
    delete[] outputs;
    for (int i = 0; i <= LAYERS; i ++) {
        hipFree (data [i]);
    }
    delete[] data;
    delete[] mappings;
}