#include "hip/hip_runtime.h"
#include "model.h"
#include "params.h"
#include "reader.h"
#include <cmath>
#include <thread>
#include "helpers.h"


/**
 * Declare model static variables
 */
Model::Layer* Model::layers;
int Model::current_layer_count;


/**
 * Initialize the general model
 */
void Model::init () {
    layers = new Layer [LAYERS];
    current_layer_count = 0;
}

#include<iostream>
/**
 * Add a new layer to the model
 * Takes number of neurons in this layers along with their weights and biases
 */
void Model::add_layer (int neuron_count, double* weights, double* biases) {
    if (current_layer_count == LAYERS) {
        hipDeviceSynchronize ();
        return;
    }
    Layer& c_layer = layers [current_layer_count];
    
    // Initialize layer
    c_layer.neuron_count = neuron_count;
    c_layer.input_count = current_layer_count > 0 ? layers [current_layer_count - 1].neuron_count : INPUT;
    current_layer_count ++;
    
    // Allocate space and copy data to device
    int b_weights = sizeof (double) * c_layer.input_count * c_layer.neuron_count;
    int b_biases = sizeof (double) * c_layer.neuron_count;
    hipMalloc (&(c_layer.weights), b_weights);
    hipMalloc (&(c_layer.biases), b_biases);
    hipMemcpyAsync (c_layer.weights, weights, b_weights, hipMemcpyHostToDevice);
    hipMemcpyAsync (c_layer.biases, biases, b_biases, hipMemcpyHostToDevice);

    // Free host weights and biases
    delete[] weights;
    delete[] biases;
}


/**
 * Free memory taken by the general model
 */
void Model::free () {
    for (int i = 0; i < LAYERS; i ++) {
        hipFree (layers [i].weights);
        hipFree (layers [i].biases);
    }
    delete[] layers;
}


/**
 * Cuda kernel code to set values of a given array to 0
 */
__global__ void clear_gpu (int n, double* data) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        data [id] = 0;
    }
}


/**
 * Cuda kernel code to process a layer (perform matrix multiplication)
 */
__global__ void process_gpu (
    int batch_size, int input_count, int neuron_count,
    double* inputs, double* weights, double* outputs
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < batch_size * neuron_count) {
        for (int j = 0; j < input_count; j ++) {
            atomicAdd (
                outputs + id,
                inputs [id / neuron_count * input_count + j] *
                weights [id % neuron_count * input_count + j]
            );
        }
    }
}


/**
 * Cuda kernel code to add biases to outputs
 */
__global__ void add_bias_gpu (int batch_size, int n, double* outputs, double* biases) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        atomicAdd (outputs + id, biases [id % batch_size]);
    }
}


/**
 * Layer processing
 * Processes the given layer by taking an input to it
*/
void Model::process (int layer) {
    // Locate current layer, its inputs and outputs
    Layer& c_layer = layers [layer];
    double* input = data [layer];
    double* c_data = data [layer + 1];

    // Setup the block and grid sizes
    int block_size = 1024;
    int grid_size = (int) ceil ((double) batch_size * c_layer.neuron_count / block_size);

    // Reset all outputs
    clear_gpu <<<grid_size, block_size>>> (batch_size * c_layer.neuron_count, c_data);

    // Perform matrix multiplication
    process_gpu <<<grid_size, block_size>>> (
        batch_size, c_layer.input_count, c_layer.neuron_count, input, c_layer.weights, c_data
    );

    // Add bias to the outputs
    add_bias_gpu <<<grid_size, block_size>>> (batch_size, c_layer.neuron_count, c_data, c_layer.biases);
}


/**
 * Cuda kernel code to calculate ReLU for given array of outputs
 */
__global__ void relu_gpu (int n, double* outputs) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        outputs [id] *= (outputs [id] > 0);
    }
}


/**
 * Activate the given layer using ReLU
 */
void Model::relu (int layer) {
    double* c_data = data [layer + 1];
    int total = layers [layer].neuron_count * batch_size;
    
    // Setup the block and grid sizes
    int block_size = 1024;
    int grid_size = (int) ceil ((double) total / block_size);

    // Perform ReLU activation
    relu_gpu <<<grid_size, block_size>>> (total, c_data);
}


/**
 * Cuda kernel code to calculate exponential sums for a batch of given outputs
 */
__global__ void expsum_gpu (int n, int categories, double* outputs, double* exp_sums) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        atomicAdd (exp_sums + id / categories, exp (outputs [id]));
    }
}


/**
 * Cuda kernel code to activate the given batch of outputs with exponential sums provided
 */
__global__ void softmax_gpu (int n, int categories, double* outputs, double* exp_sums) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        outputs [id] = exp (outputs [id]) / exp_sums [id / categories];
    }
}


/**
 * Cuda kernel code to select outputs with highest probabilities
 */
__global__ void select_gpu (int batch_size, int categories, double* outputs, int* results) {
    double max;
    for (int u = 0; u < batch_size; u ++) {
        max = 0;
        for (int i = 0; i < categories; i ++) {
            if (outputs [u * categories + i] > max) {
                max = outputs [u * categories + i];
                results [u] = i;
            }
        }
    }
}


/**
 * Activate the last layer using softmax
 */
void Model::softmax () {
    // Locate the last layer and its outputs
    int categories = layers [LAYERS - 1].neuron_count;
    int total = batch_size * categories;
    double* c_data = data [LAYERS];

    // Reset all exponential sums
    clear_gpu <<<batch_size, 1>>> (batch_size, expsums);

    // Setup the block and grid sizes
    int block_size = 1024;
    int grid_size = (int) ceil ((double) total / block_size);

    // Calculate exponential sums
    expsum_gpu <<<grid_size, block_size>>> (total, categories, c_data, expsums);

    // Perform softmax activation
    softmax_gpu <<<grid_size, block_size>>> (total, categories, c_data, expsums);

    // Process results
    select_gpu <<<1, 1>>> (batch_size, categories, c_data, results);
    
    // Copy results back to host
    hipMemcpy (outputs, results, batch_size * sizeof (int), hipMemcpyDeviceToHost);
}


/**
 * Model constructor
 * Takes a batch size as a parameter
 */
Model::Model (int batch_size) {
    this -> batch_size = batch_size;
    this -> current_input = 0;
    this -> ready = 0;
    input = new double [INPUT * batch_size];
    data = new double* [LAYERS + 1];
    hipMalloc (&(data [0]), sizeof (double) * INPUT * batch_size);
    for (int i = 1; i <= LAYERS; i ++) {
        hipMalloc (&(data [i]), sizeof (double) * layers [i - 1].neuron_count * batch_size);
    }
    hipMalloc (&expsums, sizeof (double) * batch_size);
    hipMalloc (&results, sizeof (int) * batch_size);
    outputs = new int [batch_size];
    mappings = new int [batch_size];
}


/**
 * Return true if all inputs are covered, otherwise false
 */ 
bool Model::is_ready () {
    return current_input == batch_size;
}


/**
 * Read tensor into current input array
 * Takes tensor filename, a position for mapping, and a pointer to the number of free readers
 */
void Model::process_input (const std::string& filename, int pos, std::atomic_int* free_readers) {
    mappings [current_input] = pos;
    std::thread t (read_input, filename, input + current_input * INPUT, &ready, free_readers);
    t.detach ();
    (*free_readers) --;
    current_input ++;
}


/**
 * Forward pass
 * Takes an auxiliary array to store results in,
 * a queue of models to make itself available again,
 * and a sub-batch parameter (optional)
*/
void Model::forward_pass (char* aux, tq* models, int sub_batch) {
    // Save original batch size
    int original_batch = batch_size;
    if (sub_batch > 0 && sub_batch < batch_size) {
        batch_size = sub_batch;
    }

    // Wait for all inputs to be read
    while (ready != batch_size) {IDLE}

    // Copy inputs to device
    hipMemcpy (data [0], input, sizeof (double) * INPUT * batch_size, hipMemcpyHostToDevice);

    // Activate layer K-1, then process it to layer K
    for (int i = 0; i < LAYERS - 1; i ++) {
        process (i);
        relu (i);
    }

    // Process and activate the last layer and get outputs
    process (LAYERS - 1);
    softmax ();

    // Write results to the auxilary output array
    for (int i = 0; i < batch_size; i ++) {
        int res = outputs [i];
        aux [mappings [i]] = res % 2 ? char (97 + res / 2) : char (65 + res / 2);
    }

    // Restore original batch size, current input and ready count
    batch_size = original_batch;
    current_input = 0;
    ready = 0;

    // Make this model available again by adding it to the queue of available models
    models -> push (this);
}


/**
 * Model destructor
 * Frees all model outputs and data
 */
Model::~Model () {
    delete[] input;
    hipFree (expsums);
    hipFree (results);
    for (int i = 0; i <= LAYERS; i ++) {
        hipFree (data [i]);
    }
    delete[] data;
    delete[] outputs;
    delete[] mappings;
}